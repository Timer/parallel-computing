
#include <hip/hip_runtime.h>
#include <float.h>
#include <math.h>
#include <stdio.h>

__global__ void calculateAreas(const int recs, const double w, const int offset, double *areas) {
  const int index = threadIdx.x + offset;
  if (index >= recs) return;
  const double x = index * w;
  double h = 1 - x * x;
  //Detect a 0 by accounting for roundoff error.
  h = h < DBL_EPSILON ? 0 : sqrt(h);
  areas[index] = w * h;
}

void calculateArea(const int recs, double *area) {
  double *areas = (double*) malloc(recs * sizeof(double));
  if (areas == NULL) {
    fprintf(stderr, "malloc failed!\n");
    return;
  }
  double *w_areas;
  hipError_t err = hipMalloc((void**) &w_areas, (recs * sizeof(double)));
  if (err != hipSuccess) {
    fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
    return;
  }
  const int threadCount = 512, loops = ceil((double) recs / threadCount);
  const double width = 1.0 / recs;
  for (int c = 0; c < loops; ++c) {
    //kernel<<<blocks, threads>>>
    calculateAreas<<<1, threadCount>>>(recs, width, c * threadCount, w_areas);
  }
  err = hipMemcpy(areas, w_areas, recs * sizeof(double), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(err));
    return;
  }
  *area = 0;
  for (int c = 0; c < recs; ++c) {
    *area += areas[c];
  }
  *area *= 4;
  hipFree(w_areas);
  free(areas);
}
